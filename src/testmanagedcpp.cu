#include "hip/hip_runtime.h"
#include "testmanagedcpp.hpp"
#include <cstdio>

__managed__ int m;

__global__ void check_m_on_gpu(){
    printf("m is %d\n", m);
}

int main(){
    m = -1;
    set_m_to_42_in_cpp();
    hipDeviceSynchronize();
    check_m_on_gpu<<<1,1>>>();
    hipDeviceSynchronize();
}
